#include "hip/hip_runtime.h"
#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>
#include <hip/hip_runtime.h>


namespace py = pybind11;

#define BLOCL_SIZE 16

template <typename T>
__global__ void matmul_kernel(T* a, T* b, T* c, long a_width, long c_width, long c_height)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	T sum = 0;

	if (idx < c_width && idy < c_height)
	{
		T sum = 0;
		for (int k = 0; k < a_width; k++)
		{
			sum += a[idy * a_width + k] * b[k * c_width + idx];
		}
		c[idy * c_width + idx] = sum;
	}
}


template<typename T>
py::array_t<T> matmul(py::array_t<T> &array_a, py::array_t<T> &array_b) {
	const auto &buf_info_a = array_a.request();
	const auto &buf_info_b = array_b.request();
	// check shape
	if (buf_info_a.ndim != 2 || buf_info_b.ndim != 2) {
		throw std::runtime_error("Number of dimensions must be two");
	}
	// set size
	const int a_w = buf_info_a.shape[1];
	const int a_h = buf_info_a.shape[0];
	const int b_w = buf_info_b.shape[1];
	const int b_h = buf_info_b.shape[0];
	const int c_w = b_w;
	const int c_h = a_h;
	// check dimension
	if (a_w != b_h) {
		throw std::runtime_error("Dimension mismatch");
	}

	// host pointer
	T *a_host = (T *)buf_info_a.ptr;
	T *b_host = (T *)buf_info_b.ptr;
	// declare result array
	py::array_t<T> res = py::array_t<T>(a_h * b_w);
	T *c_host = (T *)res.request().ptr;

	// device pointer
	T *a_device, *b_device, *c_device;

	// malloc
	hipError_t error = hipMalloc((void**)&a_device, a_w * a_h * sizeof(T));
	if (error != hipSuccess) {
		throw std::runtime_error(hipGetErrorString(error));
	}
	error = hipMalloc((void**)&b_device, b_w * b_h * sizeof(T));
	if (error != hipSuccess) {
		throw std::runtime_error(hipGetErrorString(error));
	}
	error = hipMalloc((void**)&c_device, b_w * a_h * sizeof(T));
	if (error != hipSuccess) {
		throw std::runtime_error(hipGetErrorString(error));
	}

	// mempcy
	error = hipMemcpy(a_device, a_host, a_w * a_h * sizeof(T), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		throw std::runtime_error(hipGetErrorString(error));
	}
	error = hipMemcpy(b_device, b_host, b_w * b_h * sizeof(T), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		throw std::runtime_error(hipGetErrorString(error));
	}

	// // thread params
	dim3 dimBlock(BLOCL_SIZE, BLOCL_SIZE);
	dim3 dimGrid((c_w + dimBlock.x - 1) / dimBlock.x, (c_h + dimBlock.y - 1) / dimBlock.y);
	
	// run kernel
	matmul_kernel<T><<<dimGrid, dimBlock>>>(a_device, b_device, c_device, a_w, c_w, c_h);

	// flush
	std::flush(std::cout);

	error = hipGetLastError();
	if (error != hipSuccess) {
		throw std::runtime_error(hipGetErrorString(error));
	}
	hipDeviceSynchronize();
	error = hipMemcpy(c_host, c_device, c_w * c_h * sizeof(T), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		throw std::runtime_error(hipGetErrorString(error));
	}

	return res;
}

PYBIND11_MODULE(matmul_cuda, m)
{
	m.def("matmul", &matmul<float>);
	m.def("matmul", &matmul<double>);
	m.def("matmul", &matmul<int>);
	m.def("matmul", &matmul<long>);
}